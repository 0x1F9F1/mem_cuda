#include "hip/hip_runtime.h"
/*
    Copyright 2018 Brick

    Permission is hereby granted, free of charge, to any person obtaining a copy of this software
    and associated documentation files (the "Software"), to deal in the Software without restriction,
    including without limitation the rights to use, copy, modify, merge, publish, distribute,
    sublicense, and/or sell copies of the Software, and to permit persons to whom the Software is
    furnished to do so, subject to the following conditions:

    The above copyright notice and this permission notice shall be included in all copies or
    substantial portions of the Software.

    THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, INCLUDING
    BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
    NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM,
    DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
    OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
*/

#include "mem/cuda_pattern.h"

#include <stdexcept>

#define check(ans) do { assert_((ans), __FILE__, __LINE__); } while (false)
inline void assert_(hipError_t code, const char *file, int line)
{
    if (code != hipSuccess)
    {
        char buffer[1024];

        snprintf(buffer, 1024, "CUDA Check Failed: %s : %s : %i", hipGetErrorString(code), file, line);

        throw std::runtime_error(buffer);
    }
}

namespace mem
{
    namespace internal
    {
        template <typename T>
        __device__ inline bool push(T* values, size_t* size, size_t max_size, T value)
        {
            size_t index = atomicAdd(size, 1);

            if (index < max_size)
            {
                values[index] = value;

                return true;
            }

            return false;
        }

        __global__ void scan_kernel(
            const byte* data,
            size_t data_length,
            const byte* bytes,
            const byte* masks,
            size_t pattern_length,
            size_t* results,
            size_t* results_count,
            size_t max_results)
        {
            size_t thread_index  = blockIdx.x * blockDim.x + threadIdx.x;
            size_t total_threads = blockDim.x * gridDim.x;

            size_t bytes_per_thread = (data_length + total_threads - 1) / total_threads;

            size_t start_index      = thread_index * bytes_per_thread;
            size_t end_index        = min(start_index + bytes_per_thread, data_length);

            const byte*       current = data + start_index;
            const byte* const end     = data + end_index;

            const size_t last = pattern_length - 1;

            for (; MEM_LIKELY(current < end); ++current)
            {
                size_t i = last;

                do
                {
                    if (MEM_LIKELY((current[i] & masks[i]) != bytes[i]))
                    {
                        goto scan_next;
                    }
                } while (MEM_LIKELY(i--));

                push(results, results_count, max_results, size_t(current - data));

            scan_next:;
            }
        }
    }

    cuda_device_data::cuda_device_data(cuda_runtime* runtime, const void* data, size_t size)
    {
        runtime->set_device();

        check(hipMalloc(&data_, size));
        check(hipMemcpy(data_, data, size, hipMemcpyHostToDevice));

        size_ = size;
    }

    cuda_device_data::~cuda_device_data()
    {
        check(hipFree(data_));
    }

    cuda_device_data::cuda_device_data(cuda_device_data&& rhs)
        : data_(rhs.data_)
        , size_(rhs.size_)
    {
        rhs.data_ = nullptr;
        rhs.size_ = 0;
    }

    cuda_runtime::cuda_runtime(int device)
        : device_(device)
    { }

    cuda_runtime::~cuda_runtime() = default;

    void cuda_runtime::set_device()
    {
        check(hipSetDevice(device_));
    }

    cuda_pattern::cuda_pattern(cuda_runtime* runtime, const pattern& pattern)
    {
        runtime->set_device();

        size_t size = pattern.size();

        check(hipMalloc(&bytes_, size));
        check(hipMalloc(&masks_, size));

        check(hipMemcpy(bytes_, pattern.bytes(), size, hipMemcpyHostToDevice));
        check(hipMemcpy(masks_, pattern.masks(), size, hipMemcpyHostToDevice));

        size_ = size;
        trimmed_size_ = pattern.trimmed_size();
    }

    cuda_pattern::~cuda_pattern()
    {
        check(hipFree(bytes_));
        check(hipFree(masks_));
    }

    std::vector<size_t> cuda_pattern::scan_all(const cuda_device_data& data, size_t max_results) const
    {
        if ((data.size() < size_) || (trimmed_size_ == 0))
        {
            return {};
        }

        size_t scan_length = (data.size() - size_) + 1;

        hipDeviceProp_t deviceProp;
        check(hipGetDeviceProperties(&deviceProp, 0));

        size_t max_threads = deviceProp.maxThreadsPerBlock;
        size_t max_blocks  = 4096;

        size_t thread_count  = min(scan_length, max_threads);
        size_t block_count   = min((scan_length + thread_count - 1) / thread_count, max_blocks);

        size_t* device_results      = nullptr;
        size_t* device_result_count = nullptr;

        check(hipMalloc((void**) &device_results, max_results * sizeof(size_t)));
        check(hipMalloc((void**) &device_result_count, sizeof(size_t)));

        const size_t zero = 0;

        check(hipMemcpy(device_result_count, &zero, sizeof(size_t), hipMemcpyHostToDevice));

        internal::scan_kernel<<<(int) block_count, (int) thread_count>>>(
            (const byte*) data.data(), scan_length,
            (const byte*) bytes_, (const byte*) masks_, trimmed_size_,
            device_results, device_result_count, max_results);

        size_t result_count = 0;

        check(hipMemcpy(&result_count, device_result_count, sizeof(size_t), hipMemcpyDeviceToHost));

        std::vector<size_t> results(min(result_count, max_results));

        check(hipMemcpy(results.data(), device_results, results.size() * sizeof(size_t), hipMemcpyDeviceToHost));

        check(hipFree(device_results));
        check(hipFree(device_result_count));

        return results;
    }
}
